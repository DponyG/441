
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <limits.h>

#define N 8
#define THREADS 8

__global__ void findLowest(int numToMinimize, int a[], int *cudaResult ){
    
    int low = threadIdx.x * numToMinimize;
    int high = low + numToMinimize -1;
    *cudaResult = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < *cudaResult){
            *cudaResult = a[i];
        }
    }
}

int main(){
    int *a;
    int min = INT_MAX;
    int low, high, cudaResult;
    int *dev_c;

    a = (int *) malloc(sizeof(int)*N);

    for(unsigned int i = 0; i < N; i++)
            a[i] = rand() % 100000;

    min = a[0];
    
    int numToMinimize = N / THREADS;
   
    hipMalloc((void**)&dev_c, sizeof(int));
    findLowest<<<1,8>>>(numToMinimize, a, dev_c);
    hipMemcpy(&cudaResult, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if(min > cudaResult){
        min = cudaResult;
    }

    printf("%d \n", min); 
}

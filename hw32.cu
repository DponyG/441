
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <limits.h>

#define N 8000000
#define THREADS 8

__global__ void findLowest(int numToMinimize, int *a, int *cudaResult ){
    
    int low = threadIdx.x * numToMinimize;
    int high = low + numToMinimize -1;
    int min = a[low];
    for (unsigned int i = low; i < high; i++){
        if(a[i] < min){
            min = a[i];
        }
    }
    cudaResult[threadIdx.x] = min;
    printf("Thread %d returned: %d \n", threadIdx.x, min);
}

int main(){
    int *a;
    int *cudaResult;
    int min = INT_MAX;
    int testMin = INT_MAX;
    int *dev_result;
    int *dev_a;

    a = (int *) malloc(sizeof(int)*N);
    cudaResult = (int *) malloc(sizeof(int)*THREADS);

    for(unsigned int i = 0; i < N; i++){
        a[i] = rand() % 100000;
        if (testMin > a[i]){
            testMin = a[i];
        } 
    }

    printf("The minimum value is: %d \n", testMin);

    int numToMinimize = N / THREADS;
   
    hipMalloc((void**)&dev_result, N*sizeof(int));
    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_result, cudaResult, THREADS*sizeof(int), hipMemcpyHostToDevice);
    findLowest<<<1,8>>>(numToMinimize, dev_a, dev_result);
    hipMemcpy(cudaResult, dev_result, THREADS*sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned int i = 0; i < THREADS; i++){
        if(min > cudaResult[i]) {
            min = cudaResult[i];
        }
    }

    hipFree(dev_result);
    hipFree(dev_a);

    printf("The Cuda Value is %d \n", min); 
}

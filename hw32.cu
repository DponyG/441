
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <limits.h>

#define N 100
#define THREADS 8

__global__ void findLowest(int numToMinimize, int *a, int *cudaResult ){
    
    int low = threadIdx.x * numToMinimize;
    int high = low + numToMinimize -1;
    int min = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < min){
            min = a[i];
        }
    }
    cudaResult[threadIdx.x] = min;
    printf("Thread %d returned: %d \n", threadIdx.x, min);
}

int main(){
    int *a;
    int *cudaResult;
    int min = INT_MAX;
    int testMin = INT_MAX;
    int *dev_result;
    int *dev_a;

    a = (int *) malloc(sizeof(int)*N);
    cudaResult = (int *) malloc(sizeof(int)*N);

    for(unsigned int i = 0; i < N; i++){
        a[i] = rand() % 100000;
        if (testMin > a[i]){
            testMin = a[i];
        } 
    }

    printf("The minimum value is: %d \n", testMin);

    int numToMinimize = N / THREADS;
   
    hipMalloc((void**)&dev_result, N*sizeof(int));
    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_result, cudaResult, N*sizeof(int), hipMemcpyHostToDevice);
    findLowest<<<1,8>>>(numToMinimize, dev_a, dev_result);
    hipMemcpy(cudaResult, dev_result, N*sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned int i = 0; i < THREADS; i++){
        if(min > cudaResult[i]) {
            min = cudaResult[i];
        }
    }

    printf("The Cuda Value is %d \n", min); 
}

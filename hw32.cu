#include "hip/hip_runtime.h"
#include "stdio.h"

#define N 8
#define THREADS 8

__global__ int findLowest(int low, int high, int a[], int *cudaResult ){
    *result = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < lowestNumber){
            lowestNumber = a[i];
        }
    }
    return *result;
}

int main(){
    int *a;
    int low, high, cudaResult, min;
    int *dev_c;

    a = (int *) malloc(sizeof(int)*N);

    for(i = 0; i < N; i++)
            a[i] = rand() % 100000;

    min = a[0];
    
    int numToMinimize = N / THREADS;
    low = rank * numToMinimize;
    high = low + numToMinimize -1;

    hipMalloc((void**)&dev_c, sizeof(int));
    findLowest<<<1,8>>>(low, high, a, dev_c);
    hipMemcpy(&cudaResult, dev_c, sizeof(int), hipMemcpyDeviceToHost )
    if(min > cudaResult){
        min = cudaResult;
    }

    printf("%d \n", min); 
}

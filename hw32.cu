#include "hip/hip_runtime.h"
#include "stdio.h"
#include <limits.h>

#define N 8
#define THREADS 8

__global__ void findLowest(int low, int high, int a[], int *cudaResult ){
    *cudaResult = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < *cudaResult){
            *cudaResult = a[i];
        }
    }
}

int main(){
    int *a;
    int min = INT_MAX;
    int low, high, cudaResult;
    int *dev_c;

    a = (int *) malloc(sizeof(int)*N);

    for(unsigned int i = 0; i < N; i++)
            a[i] = rand() % 100000;

    min = a[0];
    
    int numToMinimize = N / THREADS;
    low = threadIdx.x * numToMinimize;
    high = low + numToMinimize -1;

    hipMalloc((void**)&dev_c, sizeof(int));
    findLowest<<<1,8>>>(low, high, a, dev_c);
    hipMemcpy(&cudaResult, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if(min > cudaResult){
        min = cudaResult;
    }

    printf("%d \n", min); 
}

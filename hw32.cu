
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <limits.h>

#define N 8
#define THREADS 8

__global__ void findLowest(int numToMinimize, int a[], int *cudaResult ){
    
    int low = threadIdx.x * numToMinimize;
    int high = low + numToMinimize -1;
    *cudaResult = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < *cudaResult){
            *cudaResult = a[i];
        }
    }
}

int main(){
    int *a;
    int min = INT_MAX;
    int testMin = INT_MAX;
    int cudaResult;
    int *dev_c;

    a = (int *) malloc(sizeof(int)*N);

    for(unsigned int i = 0; i < N; i++){
        a[i] = rand() % 100000;
        if (testMin > a[i]){
            testMin = a[i];
        } 
    }

    printf("The minimum value is: %d \n", testMin);

    
    int numToMinimize = N / THREADS;
   
    hipMalloc((void**)&dev_c, sizeof(int));
    findLowest<<<1,8>>>(numToMinimize, a, dev_c);
    hipMemcpy(&cudaResult, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    if(min > cudaResult){
        min = cudaResult;
    }

    printf("The Cuda Value %d \n", min); 
}

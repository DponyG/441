#include "hip/hip_runtime.h"
#include "stdio.h"
#include <limits.h>

#define N 8
#define THREADS 8

__global__ int findLowest(int low, int high, int a[], int *cudaResult ){
    *cudaResult = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < lowestNumber){
            lowestNumber = a[i];
        }
    }
    return *cudaResult;
}

int main(){
    int *a;
    int min = INT_MAX;
    int low, high, cudaResult;
    int *dev_c;

    a = (int *) malloc(sizeof(int)*N);

    for(i = 0; i < N; i++)
            a[i] = rand() % 100000;

    min = a[0];
    
    int numToMinimize = N / THREADS;
    low = rank * numToMinimize;
    high = low + numToMinimize -1;

    hipMalloc((void**)&dev_c, sizeof(int));
    findLowest<<<1,8>>>(low, high, a, dev_c);
    hipMemcpy(&cudaResult, dev_c, sizeof(int), hipMemcpyDeviceToHost )
    if(min > cudaResult){
        min = cudaResult;
    }

    printf("%d \n", min); 
}

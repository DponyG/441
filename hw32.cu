
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <limits.h>

#define N 8
#define THREADS 8

__global__ void findLowest(int numToMinimize, int *a, int *cudaResult ){
    
    int low = threadIdx.x * numToMinimize;
    int high = low + numToMinimize -1;
    *cudaResult = a[low];
    for (int i = low; i < high; i++){
        if(a[i] < *cudaResult){
            *cudaResult = a[i];
        }
    }
    printf("%d \n", *cudaResult);
}

int main(){
    int *a;
    int min = INT_MAX;
    int testMin = INT_MAX;
    int cudaResult;
    int *dev_result;
    int *dev_a;

    a = (int *) malloc(sizeof(int)*N);

    for(unsigned int i = 0; i < N; i++){
        a[i] = rand() % 100000;
        if (testMin > a[i]){
            testMin = a[i];
        } 
    }

    printf("The minimum value is: %d \n", testMin);

    int numToMinimize = N / THREADS;
   
    hipMalloc((void**)&dev_result, sizeof(int));
    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    findLowest<<<1,8>>>(numToMinimize, dev_a, dev_result);
    hipMemcpy(&cudaResult, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    if(min > cudaResult){
        min = cudaResult;
    }

    printf("The Cuda Value is %d \n", min); 
}



#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 4
#define ROWS 3

__global__ void add(int * a, int*b)  {

    int x = threadIdx.x;
    int sum = 0;

    for(unsigned int i = 0; i < ROWS; i++){
        sum += a[i*COLUMNS+x];
    }

    b[x] = sum;
    
}

int main() {
    int a[ROWS][COLUMNS], b[COLUMNS];
    int *dev_a;
    int *dev_b;
    int sum = 0;
    int cudaSum = 0; 

    hipMalloc((void **)&dev_a, ROWS*COLUMNS*sizeof(int));
    hipMalloc((void **)&dev_b, COLUMNS*sizeof(int));


    for (int y = 0; y< ROWS; y++)
        for(int x = 0; x < COLUMNS; x++){
            a[y][x] = x;
            sum += x;
        }

    printf("The exact sum is: %d \n", sum);
    
    hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, COLUMNS*sizeof(int), hipMemcpyHostToDevice);

    add<<<1,COLUMNS>>>(dev_a, dev_b);

    hipMemcpy(b, dev_b, COLUMNS*sizeof(int), hipMemcpyDeviceToHost);
    
    for(unsigned int i = 0; i < COLUMNS; i++){
        cudaSum += b[i];
    }

    printf("The cuda sum is: %d \n", cudaSum);

    
    

}
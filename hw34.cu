

#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 8
#define ROWS 8

__global__ void add(int * a, int*b)  {

    int cacheIndex = threadIdx.x;
    

    int i = blockDim.x/2;
    while(i > 0){
        if(cacheIndex < i){
            a[blockIdx.x*COLUMNS+cacheIndex] += a[blockIdx.x*COLUMNS+cacheIndex + i];
        }
        __syncthreads();
        i/=2;
    }
    if(threadIdx.x == 0) {
        b[blockIdx.x] = a[blockIdx.x*COLUMNS];
    }
}

int main() {
    int a[ROWS][COLUMNS], b[COLUMNS];
    int *dev_a;
    int *dev_b;
    int sum = 0;
    int cudaSum = 0; 

    hipMalloc((void **)&dev_a, ROWS*COLUMNS*sizeof(int));
    hipMalloc((void **)&dev_b, COLUMNS*sizeof(int));


    for (int y = 0; y< ROWS; y++)
        for(int x = 0; x < COLUMNS; x++){
            a[y][x] = x+y;
            sum += a[y][x];
        }

    printf("The exact sum is: %d \n", sum);
    
    hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, COLUMNS*sizeof(int), hipMemcpyHostToDevice);

    add<<<8,8>>>(dev_a, dev_b);

    hipMemcpy(b, dev_b, COLUMNS*sizeof(int), hipMemcpyDeviceToHost);
    
    for(unsigned int i = 0; i < COLUMNS; i++){
        cudaSum += b[i];
    }

    printf("The cuda sum is: %d \n", cudaSum);

    hipFree(dev_a);
    hipFree(dev_b);

    
    

}
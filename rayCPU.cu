#include "hip/hip_runtime.h"
/*  Duane Shaffer
    CS 443 Spring 2019 
    Dr. Kenrick Mock
    Problem 5 - This is a simple ray tracer that shoots rays top down toward randomly generated spheres and 
                draws the sphere in a random color based on where the ray hits it.
    Based off code given by Dr. Mock. My task is to accelerate it using the GPU
    Time without acceleration: 3.907 sec
    Time with cuda acceleration: 
*/

#include "FreeImage.h"
#include "stdio.h"

#define DIM 256
#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
                            float   r,b,g;
                            float   radius;
                            float   x,y,z;
                            // Tells us if a ray hits the sphere; return the
                            // depth of the hit, or -infinity if the ray misses the sphere
                            __device__ __host__ float hit( float ox, float oy, float *n ) 
                            {
                                float dx = ox - x;
                                float dy = oy - y;
                                if (dx*dx + dy*dy < radius*radius)
                                {
                                    float dz = sqrtf( radius*radius - dx*dx - dy*dy );
                                    *n = dz / sqrtf( radius * radius );
                                    return dz + z;
                                }
                                return -INF;
                            }
                        };

#define SPHERES 80


__global__ void gpu_drawSpheres(Sphere *spheres, char *red, char *green, char *blue) {
    // for (int x = 0; x < DIM; x++) { // x = blockIdx.x switch with y
        // for (int y = 0; y < DIM; y++) { // y = threadIdx.x
            float   ox = (blockIdx.x - DIM/2);
            float   oy = (threadIdx.x - DIM/2);

            float   r=0, g=0, b=0;
            float   maxz = -INF;
	        for(int i=0; i<SPHERES; i++) {
                float   n;
                float   t = spheres[i].hit( ox, oy, &n );
                if (t > maxz) {
			        // Scale RGB color based on z depth of sphere
            		float fscale = n;
            		r = spheres[i].r * fscale;
            		g = spheres[i].g * fscale;
            		b = spheres[i].b * fscale;
            		maxz = t;
        	    }
            }
            int offset = threadIdx.x + blockIdx.x * DIM;
            // printf("%d, %d, offset: %d\n", threadIdx.x, blockIdx.x, offset);
            red[offset] = (char) (r * 255);
            green[offset] = (char) (g * 255);
            blue[offset] = (char) (b * 255);
        // }
    // }
}

// Loops through each pixel in the image (represented by arrays of
// red, green, and blue) and then for each pixel checks if a ray from
// top down hits one of the randomly generated spheres.
// If so, calculate a shade of color based on where the ray hits it.
void drawSpheres(Sphere spheres[], char *red, char *green, char *blue){
    Sphere *dev_spheres;
    char *dev_red;
    char *dev_green;
    char *dev_blue;
    hipMalloc((void**) &dev_spheres, SPHERES*sizeof(Sphere));
    hipMalloc((void**) &dev_red, DIM*DIM*sizeof(char));
    hipMalloc((void**) &dev_green, DIM*DIM*sizeof(char));
    hipMalloc((void**) &dev_blue, DIM*DIM*sizeof(char));
    hipMemcpy(dev_spheres, spheres, SPHERES*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_red, red, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_green, green, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_blue, blue, DIM*DIM*sizeof(char), hipMemcpyHostToDevice);

    dim3 blocks(2048, 1);
    dim3 grids(2048, 1)

    gpu_drawSpheres<<<grids, blocks,1>>>(dev_spheres, dev_red, dev_green, dev_blue);


    hipMemcpy(red, dev_red,DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(green, dev_green, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(blue, dev_blue, DIM*DIM*sizeof(char), hipMemcpyDeviceToHost);
    hipFree(dev_spheres);
    hipFree(dev_red);
    hipFree(dev_blue);
    hipFree(dev_green);
    
}

int main() {
  FreeImage_Initialise();
  atexit(FreeImage_DeInitialise);
  FIBITMAP * bitmap = FreeImage_Allocate(DIM, DIM, 24);
  srand(time(NULL));

  char *red;
  char *green;
  char *blue;

  // Dynamically create enough memory for DIM * DIM array of char.
  // By making these dynamic rather than auto (e.g. char red[DIM][DIM])
  // we can make them much bigger since they are allocated off the heap
  red = (char *) malloc(DIM*DIM*sizeof(char));
  green = (char *) malloc(DIM*DIM*sizeof(char));
  blue = (char *) malloc(DIM*DIM*sizeof(char));

  // Create random spheres at different coordinates, colors, radius
  Sphere spheres[SPHERES];
  for (int i = 0; i<SPHERES; i++) {
        spheres[i].r = rnd( 1.0f );
        spheres[i].g = rnd( 1.0f );
        spheres[i].b = rnd( 1.0f );
        spheres[i].x = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].y = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].z = rnd( (float) DIM ) - (DIM/2.0);
        spheres[i].radius = rnd( 200.0f ) + 40;
  }
  drawSpheres(spheres, red, green, blue);

  RGBQUAD color;
  for (int i = 0; i < DIM; i++)
  {
    for (int j = 0; j < DIM; j++)
    {
      int index = j*DIM + i;
      color.rgbRed = red[index];
      color.rgbGreen = green[index];
      color.rgbBlue = blue[index];
      FreeImage_SetPixelColor(bitmap, i, j, &color);
    }
  }
	
  FreeImage_Save(FIF_PNG, bitmap, "ray.png", 0);
  FreeImage_Unload(bitmap);
  free(red);
  free(green);
  free(blue);

  return 0;
}